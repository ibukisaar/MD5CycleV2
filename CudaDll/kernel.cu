#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <cinttypes>
#include <algorithm>

#define API_EXPORT extern "C" __declspec(dllexport) __host__

template<typename T> constexpr T Mask = 0x00'00'00'ff;

struct alignas(16) hash_t {
    union {
        struct { uint32_t a, b, c, d; };
        uint32_t hash[4];
        struct { uint64_t l0, l1; };
    };

    __device__ __host__ hash_t() = default;
    __device__ __host__ hash_t(uint32_t a, uint32_t b, uint32_t c, uint32_t d) : a(a), b(b), c(c), d(d) {}

    //__device__ __host__ hash_t() : a(0), b(0), c(0), d(0) {}
    //__device__ __host__ hash_t(uint32_t v) : a(v), b(0), c(0), d(0) {}

    __device__ __host__ bool operator==(const hash_t& other) const {
        return l0 == other.l0 && l1 == other.l1;
    }

    __device__ __host__ bool operator!=(const hash_t& other) const {
        return l0 != other.l0 || l1 != other.l1;
    }

    __device__ __host__ hash_t mask() const {
        hash_t r;
        r.a = a;
        r.b = b;
        r.c = c;
        r.d = d & Mask<uint32_t>;
        return r;
    }
};

template<size_t N>
struct alignas(4 * N) vec_t {
    uint32_t v[N];

    __device__ __host__ constexpr vec_t() = default;

    __device__ __host__ constexpr vec_t(uint32_t v) : v{} {
#pragma unroll
        for (size_t i = 0; i < N; i++) {
            this->v[i] = v;
        }
    }

    __device__ __host__ constexpr uint32_t& operator[](size_t i) {
        return v[i];
    }

    __device__ __host__ constexpr uint32_t operator[](size_t i) const {
        return v[i];
    }

    __device__ __host__ constexpr vec_t operator+(const vec_t& other) const {
        vec_t r;
#pragma unroll
        for (size_t i = 0; i < N; i++) r[i] = v[i] + other[i];
        return r;
    }

    __device__ __host__ constexpr vec_t operator&(const vec_t& other) const {
        vec_t r;
#pragma unroll
        for (size_t i = 0; i < N; i++) r[i] = v[i] & other[i];
        return r;
    }

    __device__ __host__ constexpr vec_t operator^(const vec_t& other) const {
        vec_t r;
#pragma unroll
        for (size_t i = 0; i < N; i++) r[i] = v[i] ^ other[i];
        return r;
    }

    __device__ __host__ constexpr vec_t operator|(const vec_t& other) const {
        vec_t r;
#pragma unroll
        for (size_t i = 0; i < N; i++) r[i] = v[i] | other[i];
        return r;
    }

    __device__ __host__ constexpr vec_t operator~() const {
        vec_t r;
#pragma unroll
        for (size_t i = 0; i < N; i++) r[i] = ~v[i];
        return r;
    }

    __device__ __host__ constexpr vec_t RL(int shift) const {
        vec_t r;
#pragma unroll
        for (size_t i = 0; i < N; i++) {
            r[i] = (v[i] << shift) | (v[i] >> (32 - shift));
        }
        return r;
    }
};

constexpr size_t V = 2;

struct alignas(64) hash_vec_t {
    using vec_t = ::vec_t<V>;

    union {
        struct { vec_t a, b, c, d; };
        vec_t hash[4];
    };

    __device__ __host__ hash_t operator[](size_t i) const {
        hash_t h;
        h.a = a[i];
        h.b = b[i];
        h.c = c[i];
        h.d = d[i];
        return h;
    }
};

template<size_t N> __device__ __forceinline static vec_t<N> RL(const vec_t<N>& x, int n) { return x.RL(n); }

template<typename T> __device__ __forceinline static T F(T x, T y, T z) { return (x & (y ^ z)) ^ z; }
template<typename T> __device__ __forceinline static T G(T x, T y, T z) { return (z & (x ^ y)) ^ y; }
template<typename T> __device__ __forceinline static T H(T x, T y, T z) { return x ^ y ^ z; }
template<typename T> __device__ __forceinline static T I(T x, T y, T z) { return y ^ (x | ~z); }
template<typename T> __device__ __forceinline static T RL(T x, int n) { return (x << n) | (x >> (32 - n)); }

#define R(f, a, b, c, d, m, k, s) \
    a = a + (f(b, c, d) + m + k); \
    a = RL(a, s) + b;



template<typename T, bool UseMask = false>
__device__ static void md5(T hash[4]) {
    T M[16]{};
    M[0] = hash[0];
    M[1] = hash[1];
    M[2] = hash[2];
    M[3] = UseMask ? (hash[3] & Mask<T>) : hash[3];
    M[4] = 0x80;
    M[14] = 128;

    constexpr T A = 0x67452301u, B = 0xefcdab89u, C = 0x98badcfeu, D = 0x10325476u;

    T a = A;
    T b = B;
    T c = C;
    T d = D;

    R(F, a, b, c, d, M[0], 0xd76aa478, 7);
    R(F, d, a, b, c, M[1], 0xe8c7b756, 12);
    R(F, c, d, a, b, M[2], 0x242070db, 17);
    R(F, b, c, d, a, M[3], 0xc1bdceee, 22);
    R(F, a, b, c, d, M[4], 0xf57c0faf, 7);
    R(F, d, a, b, c, M[5], 0x4787c62a, 12);
    R(F, c, d, a, b, M[6], 0xa8304613, 17);
    R(F, b, c, d, a, M[7], 0xfd469501, 22);
    R(F, a, b, c, d, M[8], 0x698098d8, 7);
    R(F, d, a, b, c, M[9], 0x8b44f7af, 12);
    R(F, c, d, a, b, M[10], 0xffff5bb1, 17);
    R(F, b, c, d, a, M[11], 0x895cd7be, 22);
    R(F, a, b, c, d, M[12], 0x6b901122, 7);
    R(F, d, a, b, c, M[13], 0xfd987193, 12);
    R(F, c, d, a, b, M[14], 0xa679438e, 17);
    R(F, b, c, d, a, M[15], 0x49b40821, 22);

    R(G, a, b, c, d, M[1], 0xf61e2562, 5);
    R(G, d, a, b, c, M[6], 0xc040b340, 9);
    R(G, c, d, a, b, M[11], 0x265e5a51, 14);
    R(G, b, c, d, a, M[0], 0xe9b6c7aa, 20);
    R(G, a, b, c, d, M[5], 0xd62f105d, 5);
    R(G, d, a, b, c, M[10], 0x02441453, 9);
    R(G, c, d, a, b, M[15], 0xd8a1e681, 14);
    R(G, b, c, d, a, M[4], 0xe7d3fbc8, 20);
    R(G, a, b, c, d, M[9], 0x21e1cde6, 5);
    R(G, d, a, b, c, M[14], 0xc33707d6, 9);
    R(G, c, d, a, b, M[3], 0xf4d50d87, 14);
    R(G, b, c, d, a, M[8], 0x455a14ed, 20);
    R(G, a, b, c, d, M[13], 0xa9e3e905, 5);
    R(G, d, a, b, c, M[2], 0xfcefa3f8, 9);
    R(G, c, d, a, b, M[7], 0x676f02d9, 14);
    R(G, b, c, d, a, M[12], 0x8d2a4c8a, 20);

    R(H, a, b, c, d, M[5], 0xfffa3942, 4);
    R(H, d, a, b, c, M[8], 0x8771f681, 11);
    R(H, c, d, a, b, M[11], 0x6d9d6122, 16);
    R(H, b, c, d, a, M[14], 0xfde5380c, 23);
    R(H, a, b, c, d, M[1], 0xa4beea44, 4);
    R(H, d, a, b, c, M[4], 0x4bdecfa9, 11);
    R(H, c, d, a, b, M[7], 0xf6bb4b60, 16);
    R(H, b, c, d, a, M[10], 0xbebfbc70, 23);
    R(H, a, b, c, d, M[13], 0x289b7ec6, 4);
    R(H, d, a, b, c, M[0], 0xeaa127fa, 11);
    R(H, c, d, a, b, M[3], 0xd4ef3085, 16);
    R(H, b, c, d, a, M[6], 0x04881d05, 23);
    R(H, a, b, c, d, M[9], 0xd9d4d039, 4);
    R(H, d, a, b, c, M[12], 0xe6db99e5, 11);
    R(H, c, d, a, b, M[15], 0x1fa27cf8, 16);
    R(H, b, c, d, a, M[2], 0xc4ac5665, 23);

    R(I, a, b, c, d, M[0], 0xf4292244, 6);
    R(I, d, a, b, c, M[7], 0x432aff97, 10);
    R(I, c, d, a, b, M[14], 0xab9423a7, 15);
    R(I, b, c, d, a, M[5], 0xfc93a039, 21);
    R(I, a, b, c, d, M[12], 0x655b59c3, 6);
    R(I, d, a, b, c, M[3], 0x8f0ccc92, 10);
    R(I, c, d, a, b, M[10], 0xffeff47d, 15);
    R(I, b, c, d, a, M[1], 0x85845dd1, 21);
    R(I, a, b, c, d, M[8], 0x6fa87e4f, 6);
    R(I, d, a, b, c, M[15], 0xfe2ce6e0, 10);
    R(I, c, d, a, b, M[6], 0xa3014314, 15);
    R(I, b, c, d, a, M[13], 0x4e0811a1, 21);
    R(I, a, b, c, d, M[4], 0xf7537e82, 6);
    R(I, d, a, b, c, M[11], 0xbd3af235, 10);
    R(I, c, d, a, b, M[2], 0x2ad7d2bb, 15);
    R(I, b, c, d, a, M[9], 0xeb86d391, 21);

    hash[0] = a + A;
    hash[1] = b + B;
    hash[2] = c + C;
    hash[3] = d + D;
}

__device__ static uint8_t hex2char(uint8_t x) {
    //constexpr uint64_t L = 0x37'36'35'34'33'32'31'30;
    //constexpr uint64_t H = 0x66'65'64'63'62'61'39'38;
    return x > 9 ? 'a' + x - 10 : '0' + x;
    //uint8_t a = x / 10;
    //uint8_t b = x % 10;
    //return a ? b + 97 : b + 48;
}

__device__ static uint16_t hex2str(uint8_t x) {
    uint8_t r[2];
    r[0] = hex2char(x >> 4);
    r[1] = hex2char(x & 15);
    return *reinterpret_cast<uint16_t*>(r);
}

__device__ static uint32_t hex2str(uint16_t x, const uint16_t* cache) {
    uint8_t r[sizeof(uint32_t)];
    for (int i = 0; i < 2; i++) {
        uint8_t b = x >> (i * 8);
        reinterpret_cast<uint16_t*>(r)[i] = cache[b];
    }
    return *reinterpret_cast<uint32_t*>(r);
}

__device__ static uint64_t hex2str(uint32_t x, const uint16_t* cache) {
    uint8_t r[sizeof(uint64_t)];
    for (int i = 0; i < 4; i++) {
        uint8_t b = x >> (i * 8);
        reinterpret_cast<uint16_t*>(r)[i] = cache[b];
    }
    return *reinterpret_cast<uint64_t*>(r);
}

__device__ __forceinline static uint64_t md5_114514(const uint64_t* prefix, uint64_t hash_d) {
    uint32_t M[16]{};
    //M[0] = '5411';
    //M[1] = '9141';
    //M[2] = '1891';
    //M[3] = '0' | (hex2str((uint16_t)hash[0]) << 8); // 3
    //*reinterpret_cast<uint64_t*>(&M[4]) = hex2str(hash[2]); // 8
    //*reinterpret_cast<uint64_t*>(&M[6]) = hex2str(hash[3]); // 8
    memcpy(M + 0, prefix, 24);
    memcpy(M + 6, &hash_d, sizeof(uint64_t));
    //*reinterpret_cast<uint64_t*>(&M[6]) = hash_d;
    M[8] = 0x80;
    M[14] = 256;

    constexpr uint32_t A = 0x67452301u, B = 0xefcdab89u, C = 0x98badcfeu, D = 0x10325476u;

    uint32_t a = A;
    uint32_t b = B;
    uint32_t c = C;
    uint32_t d = D;

    R(F, a, b, c, d, M[0], 0xd76aa478, 7);
    R(F, d, a, b, c, M[1], 0xe8c7b756, 12);
    R(F, c, d, a, b, M[2], 0x242070db, 17);
    R(F, b, c, d, a, M[3], 0xc1bdceee, 22);
    R(F, a, b, c, d, M[4], 0xf57c0faf, 7);
    R(F, d, a, b, c, M[5], 0x4787c62a, 12);
    R(F, c, d, a, b, M[6], 0xa8304613, 17);
    R(F, b, c, d, a, M[7], 0xfd469501, 22);
    R(F, a, b, c, d, M[8], 0x698098d8, 7);
    R(F, d, a, b, c, M[9], 0x8b44f7af, 12);
    R(F, c, d, a, b, M[10], 0xffff5bb1, 17);
    R(F, b, c, d, a, M[11], 0x895cd7be, 22);
    R(F, a, b, c, d, M[12], 0x6b901122, 7);
    R(F, d, a, b, c, M[13], 0xfd987193, 12);
    R(F, c, d, a, b, M[14], 0xa679438e, 17);
    R(F, b, c, d, a, M[15], 0x49b40821, 22);

    R(G, a, b, c, d, M[1], 0xf61e2562, 5);
    R(G, d, a, b, c, M[6], 0xc040b340, 9);
    R(G, c, d, a, b, M[11], 0x265e5a51, 14);
    R(G, b, c, d, a, M[0], 0xe9b6c7aa, 20);
    R(G, a, b, c, d, M[5], 0xd62f105d, 5);
    R(G, d, a, b, c, M[10], 0x02441453, 9);
    R(G, c, d, a, b, M[15], 0xd8a1e681, 14);
    R(G, b, c, d, a, M[4], 0xe7d3fbc8, 20);
    R(G, a, b, c, d, M[9], 0x21e1cde6, 5);
    R(G, d, a, b, c, M[14], 0xc33707d6, 9);
    R(G, c, d, a, b, M[3], 0xf4d50d87, 14);
    R(G, b, c, d, a, M[8], 0x455a14ed, 20);
    R(G, a, b, c, d, M[13], 0xa9e3e905, 5);
    R(G, d, a, b, c, M[2], 0xfcefa3f8, 9);
    R(G, c, d, a, b, M[7], 0x676f02d9, 14);
    R(G, b, c, d, a, M[12], 0x8d2a4c8a, 20);

    R(H, a, b, c, d, M[5], 0xfffa3942, 4);
    R(H, d, a, b, c, M[8], 0x8771f681, 11);
    R(H, c, d, a, b, M[11], 0x6d9d6122, 16);
    R(H, b, c, d, a, M[14], 0xfde5380c, 23);
    R(H, a, b, c, d, M[1], 0xa4beea44, 4);
    R(H, d, a, b, c, M[4], 0x4bdecfa9, 11);
    R(H, c, d, a, b, M[7], 0xf6bb4b60, 16);
    R(H, b, c, d, a, M[10], 0xbebfbc70, 23);
    R(H, a, b, c, d, M[13], 0x289b7ec6, 4);
    R(H, d, a, b, c, M[0], 0xeaa127fa, 11);
    R(H, c, d, a, b, M[3], 0xd4ef3085, 16);
    R(H, b, c, d, a, M[6], 0x04881d05, 23);
    R(H, a, b, c, d, M[9], 0xd9d4d039, 4);
    R(H, d, a, b, c, M[12], 0xe6db99e5, 11);
    R(H, c, d, a, b, M[15], 0x1fa27cf8, 16);
    R(H, b, c, d, a, M[2], 0xc4ac5665, 23);

    R(I, a, b, c, d, M[0], 0xf4292244, 6);
    R(I, d, a, b, c, M[7], 0x432aff97, 10);
    R(I, c, d, a, b, M[14], 0xab9423a7, 15);
    R(I, b, c, d, a, M[5], 0xfc93a039, 21);
    R(I, a, b, c, d, M[12], 0x655b59c3, 6);
    R(I, d, a, b, c, M[3], 0x8f0ccc92, 10);
    R(I, c, d, a, b, M[10], 0xffeff47d, 15);
    R(I, b, c, d, a, M[1], 0x85845dd1, 21);
    R(I, a, b, c, d, M[8], 0x6fa87e4f, 6);
    R(I, d, a, b, c, M[15], 0xfe2ce6e0, 10);
    R(I, c, d, a, b, M[6], 0xa3014314, 15);
    R(I, b, c, d, a, M[13], 0x4e0811a1, 21);
    R(I, a, b, c, d, M[4], 0xf7537e82, 6);
    R(I, d, a, b, c, M[11], 0xbd3af235, 10);
    R(I, c, d, a, b, M[2], 0x2ad7d2bb, 15);
    R(I, b, c, d, a, M[9], 0xeb86d391, 21);

    //hash_t r;
    //r.a = a + A;
    //r.b = b + B;
    //r.c = c + C;
    //r.d = d + D;
    //return r;

    //const uint32_t r[2]{ a + A, b + B };
    //return *reinterpret_cast<const uint64_t*>(r);
    return static_cast<uint64_t>(a + A) | (static_cast<uint64_t>(b + B) << 32);
}

struct result_t {
    union {
        hash_t __align__(4) hash;
        uint32_t index;
    };
    uint64_t iterateCnt;
};

struct result114514_t {
    hash_t __align__(4) hash;
    uint32_t index;
    uint32_t iterateCnt;
};


static int blockCount, threadCount;
static hash_t* cpu_hashes;
static result_t* cpu_result;
static result114514_t* cpu_result114514;
static int cpu_maxResultCount;

__constant__ static hash_t* hashes;
__constant__ static result_t* result;
__constant__ static result114514_t* result114514;
__constant__ static int maxResultCount;
__device__ static int resultCount;

constexpr uint64_t N = 1 << 24;

__global__ void gpu_md5(uint64_t start) {
    uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;
    hash_t h = hashes[id];

#pragma unroll 2
    for (uint64_t n = 0; n < N; n++) {
        md5<uint32_t>(h.hash);

        if (h.a == 0) { // 32 bits
            int resultIndex = atomicAdd(&resultCount, 1);
            if (resultIndex < maxResultCount) {
                result[resultIndex].hash = h;
                result[resultIndex].index = id;
                result[resultIndex].iterateCnt = start + n + 1;
            }
        }

        //__syncthreads();
    }

    hashes[id] = h;
}

__global__ void gpu_md5_mask(uint64_t start) {
    uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;
    hash_t h = hashes[id];

#pragma unroll 2
    for (uint64_t n = 0; n < N; n++) {
        md5<uint32_t, true>(h.hash);

        if (h.a == 0) { // 32 bits
            int resultIndex = atomicAdd(&resultCount, 1);
            if (resultIndex < maxResultCount) {
                result[resultIndex].hash = h.mask();
                result[resultIndex].index = id;
                result[resultIndex].iterateCnt = start + n + 1;
            }
        }

        //__syncthreads();
    }

    hashes[id] = h;
}

__global__ void gpu_md5_vec(uint64_t start) {
    uint32_t idStart = (blockDim.x * blockIdx.x + threadIdx.x) * V;
    hash_vec_t h;

    for (size_t i = 0; i < V; i++) {
        h.a[i] = hashes[idStart + i].a;
        h.b[i] = hashes[idStart + i].b;
        h.c[i] = hashes[idStart + i].c;
        h.d[i] = hashes[idStart + i].d;
    }

#pragma unroll 2
    for (uint64_t n = 0; n < N; n++) {
        md5<hash_vec_t::vec_t>(h.hash);

#pragma unroll
        for (size_t i = 0; i < V; i++) {
            if (h.a[i] == 0) {
                int resultIndex = atomicAdd(&resultCount, 1);
                if (resultIndex < maxResultCount) {
                    result[resultIndex].hash = h[i];
                    result[resultIndex].index = idStart + i;
                    result[resultIndex].iterateCnt = start + n + 1;
                }
            }
        }

        //__syncthreads();
    }

    for (size_t i = 0; i < V; i++) {
        hashes[idStart + i].a = h.a[i];
        hashes[idStart + i].b = h.b[i];
        hashes[idStart + i].c = h.c[i];
        hashes[idStart + i].d = h.d[i];
    }
}

API_EXPORT
int md5(uint64_t& start, result_t* result, int useMask) {
    int resultCount = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(::resultCount), &resultCount, sizeof(int));

    if (useMask) {
        gpu_md5_mask << <blockCount, threadCount >> > (start);
    }
    else {
        gpu_md5 << <blockCount, threadCount >> > (start);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        return -(int)error;
    }
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&resultCount, HIP_SYMBOL(::resultCount), sizeof(int));

    if (resultCount) {
        int copyCount = std::min(resultCount, cpu_maxResultCount);
        hipMemcpy(result, cpu_result, (size_t)copyCount * sizeof(result_t), hipMemcpyDeviceToHost);
    }

    start += N;
    return resultCount;
}

API_EXPORT
int md5_vec(uint64_t& start, result_t* result) {
    int resultCount = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(::resultCount), &resultCount, sizeof(int));

    gpu_md5_vec << <blockCount, (threadCount / V) >> > (start);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        return -(int)error;
    }
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&resultCount, HIP_SYMBOL(::resultCount), sizeof(int));

    if (resultCount) {
        int copyCount = std::min(resultCount, cpu_maxResultCount);
        hipMemcpy(result, cpu_result, (size_t)copyCount * sizeof(result_t), hipMemcpyDeviceToHost);
    }

    start += N;
    return resultCount;
}

__global__ void gpu_114514_md5(uint64_t start) {
    uint32_t id = blockIdx.x;
    
    __shared__ uint16_t hex_cache[256];
    __shared__ uint64_t common_prefix[3];

    if (threadIdx.x < 32) {
        for (int i = 0; i < 256; i += 32) {
            hex_cache[i + threadIdx.x] = hex2str((uint8_t)(i + threadIdx.x));
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        hash_t h = hashes[id];
        common_prefix[0] = 0x39'31'34'31'35'34'31'31;
        common_prefix[1] = 0x30'31'38'39'31 | ((uint64_t)hex2str((uint16_t)h.a, hex_cache) << 40);
        common_prefix[2] = hex2str(h.c, hex_cache);
    }

    __syncthreads();

    uint64_t prefix3 = (uint64_t)hex_cache[threadIdx.x] << 48;

    for (uint32_t i2 = 0; i2 < 256; i2++) {
        uint64_t prefix2 = prefix3 | ((uint64_t)hex_cache[i2] << 32);
        for (uint32_t i1 = 0; i1 < 256; i1++) {
            uint64_t prefix1 = prefix2 | ((uint64_t)hex_cache[i1] << 16);
            for (uint32_t i0 = 0; i0 < 256; i0++) {
                uint64_t prefix0 = prefix1 | (uint64_t)hex_cache[i0];
                uint64_t r = md5_114514(common_prefix, prefix0);

                // 0x00811919'144511
                if ((r & 0x00ffffff'ffffff) == 0x00811919'144511) {
                    int resultIndex = atomicAdd(&resultCount, 1);
                    if (resultIndex < maxResultCount) {
                        hash_t h = hashes[id];
                        result114514[resultIndex].hash = hash_t(h.a, h.b, h.c, threadIdx.x * 256 * 256 * 256 + i2 * 256 * 256 + i1 * 256 + i0);
                        //result114514[resultIndex].index = id;
                        //result114514[resultIndex].iterateCnt = start + n + 1;
                    }
                }
            }
        }
    }
}

API_EXPORT
int _114514_md5(uint64_t& start, result114514_t* result) {
    int resultCount = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(::resultCount), &resultCount, sizeof(int));

    gpu_114514_md5 << <blockCount, threadCount >> > (start);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        return -(int)error;
    }
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&resultCount, HIP_SYMBOL(::resultCount), sizeof(int));

    if (resultCount) {
        int copyCount = std::min(resultCount, cpu_maxResultCount);
        hipMemcpy(result, cpu_result114514, (size_t)copyCount * sizeof(result114514_t), hipMemcpyDeviceToHost);
    }

    start += N;
    return resultCount;
}

API_EXPORT
void read_hashes(hash_t* output) {
    hipMemcpy(output, cpu_hashes, (size_t)blockCount * threadCount * sizeof(hash_t), hipMemcpyDeviceToHost);
}

API_EXPORT
const char* get_error(int error) {
    return hipGetErrorString((hipError_t)-error);
}

API_EXPORT
void init(int blockCount, int threadCount, int maxResultCount, const hash_t* input) {
    hipMalloc(&cpu_hashes, (size_t)blockCount * threadCount * sizeof(hash_t));
    if (input) {
        hipMemcpy(cpu_hashes, input, (size_t)blockCount * threadCount * sizeof(hash_t), hipMemcpyHostToDevice);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(hashes), &cpu_hashes, sizeof(void*));

    hipMalloc(&cpu_result, (size_t)maxResultCount * sizeof(result_t));
    hipMemcpyToSymbol(HIP_SYMBOL(result), &cpu_result, sizeof(void*));

    hipMemcpyToSymbol(HIP_SYMBOL(::maxResultCount), &maxResultCount, sizeof(int));

    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    ::blockCount = blockCount;
    ::threadCount = threadCount;
    cpu_maxResultCount = maxResultCount;
}

API_EXPORT
void init114514(int blockCount, int threadCount, int maxResultCount) {
    hipMalloc(&cpu_hashes, (size_t)blockCount * sizeof(hash_t));
    hipMemcpyToSymbol(HIP_SYMBOL(hashes), &cpu_hashes, sizeof(void*));

    hipMalloc(&cpu_result114514, (size_t)maxResultCount * sizeof(result114514_t));
    hipMemcpyToSymbol(HIP_SYMBOL(result114514), &cpu_result114514, sizeof(void*));

    hipMemcpyToSymbol(HIP_SYMBOL(::maxResultCount), &maxResultCount, sizeof(int));

    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    ::blockCount = blockCount;
    ::threadCount = threadCount;
    cpu_maxResultCount = maxResultCount;
}

API_EXPORT
void write_hashes(const hash_t* input) {
    hipMemcpy(cpu_hashes, input, (size_t)blockCount * sizeof(hash_t), hipMemcpyHostToDevice);
}

API_EXPORT
void release() {
    hipFree(cpu_hashes);
    hipFree(cpu_result);
}

API_EXPORT
void release114514() {
    hipFree(cpu_hashes);
    hipFree(cpu_result114514);
}